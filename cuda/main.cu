#include "hip/hip_runtime.h"
#include <float.h>
#include "./lib/io.cpp"
#include "../lib/types.cpp"
using namespace std;

void print_cluster(int k, int *cluster_assignment,int n,const cluster_dataset& data){
  vector<int> ret[k];
  for (int i=0; i<n; i++){
    ret[cluster_assignment[i]].push_back(i);
  }
  for (int i=0; i<k; i++){
    cout << i << " ::: ";
    for (int j=0; j<ret[i].size(); j++)
      cout << data.names[ret[i][j]] << ", ";
    cout << endl << endl;
  }
}

void print_cluster_size(int k, int *cluster_assignment,int n){
  int size[k];
  for (int i=0; i<k; i++)
    size[i] = 0;
  for (int i=0; i<n; i++){
    size[cluster_assignment[i]]++;
  }
  for (int i=0; i<k; i++){
    cout << i << " ::: " << size[i] << endl;
  }
}


float euclidean_distance_array(const float * x,const float * y, int n) {
    float sum = 0;
    for (int i=0; i < n; i++) {
      sum += pow(x[i] - y[i], 2);
    }
    return sqrt(sum);
}
  

// cluster assignment using randomization
// __global__
void init_cluster_assignment(int k, int size, int * cluster_size, int * cluster_assignment){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    int group = 0;
  
    // cluster_assignment = new int[size];
    srand(static_cast<unsigned int>(clock()));
    for (int i=0; i<size; i++){
      group = (int) rand() % k;
      cluster_assignment[i] = group;
      cluster_size[group] += 1;
    }
  }

bool update_clusters(int k, float ** cluster, const int * cluster_assignment, const cluster_dataset &data, const int * cluster_size){
bool did_change = 0;

float ** temp;
temp = new float* [k];
for (int i=0; i<k; i++)
    temp[i] = new float[data.dimensions];

for (int i=0; i<k; i++){
    for (int j=0; j<data.dimensions; j++){
    temp[i][j] = (float) 0;
    }
}  

for (int i=0; i<data.size; i++){
    for (int j=0; j<data.dimensions; j++){
    temp[cluster_assignment[i]][j] += data.features[i][j];   
    }
}

for (int i=0; i<k; i++){
    if (cluster_size[i] == 0){
    cout << "ZERO ::: " << i << endl;
    continue;
    }
    for (int j=0; j<data.dimensions; j++){
    if (cluster[i][j] != temp[i][j]/cluster_size[i]){
        did_change = 1;
    }
    cluster[i][j] = temp[i][j]/cluster_size[i];
    }
}

return did_change;
}


int find_nearest_center(int k, const float * features, int dimensions,float ** cluster){
float minDist = FLT_MAX;
int minIndex = 0;
float dist = 0;
for (int i=0; i<k; i++){
    dist = euclidean_distance_array(features, cluster[i], dimensions);
    if (dist < minDist) {
        minDist = dist;
        minIndex = i;
    }
}
return minIndex;
}

void print_cluster_assignment(int k, int *cluster_assignment,int n){
vector<int> ret[k];
for (int i=0; i<n; i++){
    ret[cluster_assignment[i]].push_back(i);
}
}

void update_cluster_assignment(int k, int * cluster_assignment, int * cluster_size, float ** cluster, const cluster_dataset &data){
for (int i=0; i<k; i++){
    cluster_size[i] = 0;
}

for (int i=0; i<data.size; i++){
    cluster_assignment[i] = find_nearest_center(k, data.features[i], data.dimensions, cluster);
    cluster_size[cluster_assignment[i]]++;
}
}


int * find_clusters(int k, const cluster_dataset data, int max_iter) {
// int iter = 0;
int * cluster_size;
int * cluster_assignment;

// cluster_size = new int[k]; 

hipMallocManaged(&cluster_size, k*sizeof(int));
hipMallocManaged(&cluster_assignment, data.size*sizeof(int));

init_cluster_assignment(k, data.size, cluster_size, cluster_assignment);

    float ** cluster;
    cluster = new float* [k];
    for (int i=0; i<k; i++)
        cluster[i] = new float[data.dimensions];


    for (int i=0; i < max_iter; i++) {
        // cout << iter++ << endl;
        bool did_change = update_clusters(k, cluster, cluster_assignment, data, cluster_size);
        if (did_change){
        update_cluster_assignment(k, cluster_assignment, cluster_size, cluster, data);
        }
        else{
        return cluster_assignment;
        }
    }
    return cluster_assignment;
}



// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(){
  data_map college_data = read_csv("./datasets/College.csv");
  cluster_dataset data = fill_college_struct(college_data);
  
  int k = 10; 
  int * cluster = find_clusters(k, data, 100);

  // print_cluster(k, cluster, data.size, data);
  print_cluster_size(k, cluster, data.size);
 ///////////
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);


}