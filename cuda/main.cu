#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include "./lib/read_csv.cpp"
#include "../lib/types.cpp"
#include "./lib/helpers.cu"
using namespace std;

float euclidean_distance_array(const float * x,const float * y, int n) {
    float sum = 0;
    for (int i=0; i < n; i++) {
      sum += pow(x[i] - y[i], 2);
    }
    return sqrt(sum);
}
  

void calculate_cluster_size(int k, int *cluster_assignment,int n, int * cluster_size){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    for (int i=0; i<n; i++){
      cluster_size[cluster_assignment[i]]++;
    }
  }

// cluster assignment using randomization
__global__
void init_cluster_assignment(int k, int size, int * cluster_size, int * cluster_assignment){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    int group = 0;

    // srand(static_cast<unsigned int>(clock()));
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int random = index % k ;

    // int random = hiprandGenerate();
    // std::printf("block id : %d \t block dim: %d \t thread id : %d \t index : %d \t random : %d \n",blockIdx.x, blockDim.x, threadIdx.x, index, random);

    // cluster_assignment = new int[size];
    for (int i=index; i<size; i+=stride){
      group = (int) random;
    //   group = (int) (rand() % k);
      cluster_assignment[i] = group;
      cluster_size[group] += 1;
    }
  }

__global__
void update_clusters(int k, float ** cluster, const int * cluster_assignment, const int dimensions, float  ** feature_vector,const int * cluster_size, int * response){
    response[0] = 0;

    float ** temp;
    temp = new float* [k];
    for (int i=0; i<k; i++)
        temp[i] = new float[dimensions];

    for (int i=0; i<k; i++){
        for (int j=0; j<dimensions; j++){
        temp[i][j] = (float) 0;
        }
    }  

    for (int i=0; i<data.size; i++){
        for (int j=0; j<dimensions; j++){
        temp[cluster_assignment[i]][j] += feature_vector[i][j];   
        }
    }

    printf("College :: %d \n", dimensions);

    for (int i=0; i<k; i++){
        if (cluster_size[i] == 0){
            std::printf("ZERO ::: %d \n", i);    
        // cout << "ZERO ::: " << i << endl;
        continue;
        }
        for (int j=0; j<dimensions; j++){
        if (cluster[i][j] != temp[i][j]/cluster_size[i]){
            response[0] = 1;
        }
        cluster[i][j] = temp[i][j]/cluster_size[i];
        }
    }
}

int find_nearest_center(int k, const float * features, int dimensions,float ** cluster){
    float minDist = FLT_MAX;
    int minIndex = 0;
    float dist = 0;
    for (int i=0; i<k; i++){
        dist = euclidean_distance_array(features, cluster[i], dimensions);
        if (dist < minDist) {
            minDist = dist;
            minIndex = i;
        }
    }
    return minIndex;
}

// __global__
void update_cluster_assignment(int k, int * cluster_assignment, int * cluster_size, float ** cluster, const college_dataset &data){
    for (int i=0; i<k; i++){
        cluster_size[i] = 0;
    }

    for (int i=0; i<data.size; i++){
        cluster_assignment[i] = find_nearest_center(k, data.features[i], data.dimensions, cluster);
        cluster_size[cluster_assignment[i]]++;
    }
}

void parse_data(const data_map &data, int &size, int &dimensions, string ** data_title, float *** data_vector){
    vector<float> sample_map_data = data.begin()->second; 
    size = data.size();   
    dimensions = sample_map_data.size();

    // hipMallocManaged(&data_title, size*sizeof(string)); 
    // hipMallocManaged(&data_vector, size*sizeof(float*));
    // for (int i=0; i<size; i++)
    //     hipMallocManaged(&data_vetor[i], dimensions*sizeof(float)); 

    // int index = 0;
    // for (data_map::const_iterator it = data.begin(); it != data.end(); it++) {
    //     data_title[index] = it->first;
    //     for (int j=0; j<dimensions; j++)
    //         data_vetor[index][j] = (it->second)[j];
    //     index++;
    // } 

}


int * find_clusters(int k, const data_map data, int max_iter) {
    // int iter = 0;
    int * cluster_size;
    int * cluster_assignment;

    // college data parsing
    int data_dimensions;
    int data_size;
    string * data_title;
    float ** data_vector;
    parse_data(data, data_dimensions, data_size, &data_title, &data_vector);
    cout << "Size : " << data_size << "  dim : " << data_dimensions << endl;


    hipMallocManaged(&cluster_size, k*sizeof(int));
    hipMallocManaged(&cluster_assignment, data_size*sizeof(int));

    int blockSize = 256;
    int numBlocks = (data_size + blockSize - 1) / blockSize;

    init_cluster_assignment<<<numBlocks ,blockSize >>>(k, data_size, cluster_size, cluster_assignment);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    calculate_cluster_size(k, cluster_assignment, data_size, cluster_size);

    float ** cluster;
    hipMallocManaged(&cluster, k*sizeof(float*));

    for (int i=0; i<k; i++)
        hipMallocManaged(&cluster[i], data_dimensions*sizeof(float));

    int * did_change;
    hipMallocManaged(&did_change, sizeof(int));

    for (int i=0; i < max_iter; i++) {
        update_clusters<<<1,1>>>(k, cluster, cluster_assignment, data_dimensions, data_vector, cluster_size, did_change);
        hipDeviceSynchronize();

        // if (did_change[0] == 1){
        //     update_cluster_assignment(k, cluster_assignment, cluster_size, cluster, data);
        // }
        // update_cluster_assignment<<<numBlocks ,blockSize >>>(k, cluster_assignment, cluster_size, cluster, data);
        // hipDeviceSynchronize();
                // }
                // else{    
                //     return cluster_assignment;
                // }
            }
    return cluster_assignment;
}


// // CUDA Kernel function to add the elements of two arrays on the GPU
// __global__
// void add(int n, float *x, float *y)
// {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride)
//     y[i] =i;
// }





int main(){
  data_map data = read_csv("./datasets/College.csv");

  int k = 10; 
  int * cluster = find_clusters(k, data, 1);


//   hipFree(x);
//   hipFree(y);
}