#include "hip/hip_runtime.h"
#include <float.h>
<<<<<<< HEAD
#include "../lib/read_csv.cpp"
=======
#include "./lib/io.cpp"
>>>>>>> 081c73d7e15a80ba67712fc54785e374a78ad292
#include "../lib/types.cpp"
#include "./host/host.cu"
using namespace std;

<<<<<<< HEAD

int main(){
  data_map data = read_csv("../datasets/College.csv");

=======
void print_cluster(int k, int *cluster_assignment,int n,const cluster_dataset& data){
  vector<int> ret[k];
  for (int i=0; i<n; i++){
    ret[cluster_assignment[i]].push_back(i);
  }
  for (int i=0; i<k; i++){
    cout << i << " ::: ";
    for (int j=0; j<ret[i].size(); j++)
      cout << data.names[ret[i][j]] << ", ";
    cout << endl << endl;
  }
}

void print_cluster_size(int k, int *cluster_assignment,int n){
  int size[k];
  for (int i=0; i<k; i++)
    size[i] = 0;
  for (int i=0; i<n; i++){
    size[cluster_assignment[i]]++;
  }
  for (int i=0; i<k; i++){
    cout << i << " ::: " << size[i] << endl;
  }
}


float euclidean_distance_array(const float * x,const float * y, int n) {
    float sum = 0;
    for (int i=0; i < n; i++) {
      sum += pow(x[i] - y[i], 2);
    }
    return sqrt(sum);
}
  

// cluster assignment using randomization
// __global__
void init_cluster_assignment(int k, int size, int * cluster_size, int * cluster_assignment){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    int group = 0;
  
    // cluster_assignment = new int[size];
    srand(static_cast<unsigned int>(clock()));
    for (int i=0; i<size; i++){
      group = (int) rand() % k;
      cluster_assignment[i] = group;
      cluster_size[group] += 1;
    }
  }

bool update_clusters(int k, float ** cluster, const int * cluster_assignment, const cluster_dataset &data, const int * cluster_size){
bool did_change = 0;

float ** temp;
temp = new float* [k];
for (int i=0; i<k; i++)
    temp[i] = new float[data.dimensions];

for (int i=0; i<k; i++){
    for (int j=0; j<data.dimensions; j++){
    temp[i][j] = (float) 0;
    }
}  

for (int i=0; i<data.size; i++){
    for (int j=0; j<data.dimensions; j++){
    temp[cluster_assignment[i]][j] += data.features[i][j];   
    }
}

for (int i=0; i<k; i++){
    if (cluster_size[i] == 0){
    cout << "ZERO ::: " << i << endl;
    continue;
    }
    for (int j=0; j<data.dimensions; j++){
    if (cluster[i][j] != temp[i][j]/cluster_size[i]){
        did_change = 1;
    }
    cluster[i][j] = temp[i][j]/cluster_size[i];
    }
}

return did_change;
}


int find_nearest_center(int k, const float * features, int dimensions,float ** cluster){
float minDist = FLT_MAX;
int minIndex = 0;
float dist = 0;
for (int i=0; i<k; i++){
    dist = euclidean_distance_array(features, cluster[i], dimensions);
    if (dist < minDist) {
        minDist = dist;
        minIndex = i;
    }
}
return minIndex;
}

void print_cluster_assignment(int k, int *cluster_assignment,int n){
vector<int> ret[k];
for (int i=0; i<n; i++){
    ret[cluster_assignment[i]].push_back(i);
}
}

void update_cluster_assignment(int k, int * cluster_assignment, int * cluster_size, float ** cluster, const cluster_dataset &data){
for (int i=0; i<k; i++){
    cluster_size[i] = 0;
}

for (int i=0; i<data.size; i++){
    cluster_assignment[i] = find_nearest_center(k, data.features[i], data.dimensions, cluster);
    cluster_size[cluster_assignment[i]]++;
}
}


int * find_clusters(int k, const cluster_dataset data, int max_iter) {
// int iter = 0;
int * cluster_size;
int * cluster_assignment;

// cluster_size = new int[k]; 

hipMallocManaged(&cluster_size, k*sizeof(int));
hipMallocManaged(&cluster_assignment, data.size*sizeof(int));

init_cluster_assignment(k, data.size, cluster_size, cluster_assignment);

    float ** cluster;
    cluster = new float* [k];
    for (int i=0; i<k; i++)
        cluster[i] = new float[data.dimensions];


    for (int i=0; i < max_iter; i++) {
        // cout << iter++ << endl;
        bool did_change = update_clusters(k, cluster, cluster_assignment, data, cluster_size);
        if (did_change){
        update_cluster_assignment(k, cluster_assignment, cluster_size, cluster, data);
        }
        else{
        return cluster_assignment;
        }
    }
    return cluster_assignment;
}



// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(){
  data_map college_data = read_csv("./datasets/College.csv");
  cluster_dataset data = fill_college_struct(college_data);
  
>>>>>>> 081c73d7e15a80ba67712fc54785e374a78ad292
  int k = 10; 
  int * cluster = find_clusters(k, data, 100);

}