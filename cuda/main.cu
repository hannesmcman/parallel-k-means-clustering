#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include "./lib/read_csv.cpp"
#include "../lib/types.cpp"
#include "./lib/helpers.cu"
using namespace std;

float euclidean_distance_array(const float * x,const float * y, int n) {
    float sum = 0;
    for (int i=0; i < n; i++) {
      sum += pow(x[i] - y[i], 2);
    }
    return sqrt(sum);
}
  

void calculate_cluster_size(int k, int *cluster_assignment,int n, int * cluster_size){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    for (int i=0; i<n; i++){
      cluster_size[cluster_assignment[i]]++;
    }
  }

// cluster assignment using randomization
__global__
void init_cluster_assignment(int k, int size, int * cluster_size, int * cluster_assignment){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    int group = 0;

    // srand(static_cast<unsigned int>(clock()));
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int random = index % k ;

    // int random = hiprandGenerate();
    // std::printf("block id : %d \t block dim: %d \t thread id : %d \t index : %d \t random : %d \n",blockIdx.x, blockDim.x, threadIdx.x, index, random);

    // cluster_assignment = new int[size];
    for (int i=index; i<size; i+=stride){
      group = (int) random;
    //   group = (int) (rand() % k);
      cluster_assignment[i] = group;
      cluster_size[group] += 1;
    }
  }

bool update_clusters(int k, float ** cluster, const int * cluster_assignment, const college_dataset &data, const int * cluster_size, bool &response){
    response = 0;

    float ** temp;
    temp = new float* [k];
    for (int i=0; i<k; i++)
        temp[i] = new float[data.dimensions];

    for (int i=0; i<k; i++){
        for (int j=0; j<data.dimensions; j++){
        temp[i][j] = (float) 0;
        }
    }  

    for (int i=0; i<data.size; i++){
        for (int j=0; j<data.dimensions; j++){
        temp[cluster_assignment[i]][j] += data.features[i][j];   
        }
    }

    for (int i=0; i<k; i++){
        if (cluster_size[i] == 0){
        cout << "ZERO ::: " << i << endl;
        continue;
        }
        for (int j=0; j<data.dimensions; j++){
        if (cluster[i][j] != temp[i][j]/cluster_size[i]){
            response = 1;
        }
        cluster[i][j] = temp[i][j]/cluster_size[i];
        }
    }
}

int find_nearest_center(int k, const float * features, int dimensions,float ** cluster){
    float minDist = FLT_MAX;
    int minIndex = 0;
    float dist = 0;
    for (int i=0; i<k; i++){
        dist = euclidean_distance_array(features, cluster[i], dimensions);
        if (dist < minDist) {
            minDist = dist;
            minIndex = i;
        }
    }
    return minIndex;
}

// __global__
void update_cluster_assignment(int k, int * cluster_assignment, int * cluster_size, float ** cluster, const college_dataset &data){
    for (int i=0; i<k; i++){
        cluster_size[i] = 0;
    }

    for (int i=0; i<data.size; i++){
        cluster_assignment[i] = find_nearest_center(k, data.features[i], data.dimensions, cluster);
        cluster_size[cluster_assignment[i]]++;
    }
}


int * find_clusters(int k, const college_dataset data, int max_iter) {
    int iter = 0;
    int * cluster_size;
    int * cluster_assignment;

    // cluster_size = new int[k]; 

    hipMallocManaged(&cluster_size, k*sizeof(int));
    hipMallocManaged(&cluster_assignment, data.size*sizeof(int));

    int blockSize = 256;
    int numBlocks = (data.size + blockSize - 1) / blockSize;

    init_cluster_assignment<<<numBlocks ,blockSize >>>(k, data.size, cluster_size, cluster_assignment);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    calculate_cluster_size(k, cluster_assignment, data.size, cluster_size);

    print_cluster_size(k,cluster_assignment,data.size);

    float ** cluster;
    hipMallocManaged(&cluster, k*sizeof(float*));

    // cluster = new float* [k];
    for (int i=0; i<k; i++)
        hipMallocManaged(&cluster[i], data.dimensions*sizeof(float));
        // cluster[i] = new float[data.dimensions];

    bool did_change;
    hipMallocManaged(did_change, sizeof(int));


    for (int i=0; i < max_iter; i++) {
        cout << "iteration : " << iter++ << endl; 
        update_clusters(k, cluster, cluster_assignment, data, cluster_size, did_change);

        for (int i=0; i<k; i++)
            cout << i << ":" << cluster_size[i] << endl;

        if (did_change){
            cout << "did change : " << iter << endl;
            update_cluster_assignment(k, cluster_assignment, cluster_size, cluster, data);

        // update_cluster_assignment<<<numBlocks ,blockSize >>>(k, cluster_assignment, cluster_size, cluster, data);
        // hipDeviceSynchronize();
        }
        else{    
            print_cluster_size(k,cluster_assignment,data.size);
            return cluster_assignment;
        }
        iter++;
    }
    print_cluster_size(k, cluster_assignment, data.size);
    return cluster_assignment;
}


// // CUDA Kernel function to add the elements of two arrays on the GPU
// __global__
// void add(int n, float *x, float *y)
// {
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for (int i = index; i < n; i += stride)
//     y[i] =i;
// }





int main(){
  data_map college_data = read_csv("./datasets/College.csv");
  college_dataset data = fill_college_struct(college_data);
  
  int k = 10; 
  int * cluster = find_clusters(k, data, 100);

  print_cluster_size(k, cluster, data.size);
//   int N = 1<<20;
//   float *x, *y;

//   // Allocate Unified Memory – accessible from CPU or GPU
//   hipMallocManaged(&x, N*sizeof(float));
//   hipMallocManaged(&y, N*sizeof(float));

//   // initialize x and y arrays on the host
//   for (int i = 0; i < N; i++) {
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//   }

//   // Run kernel on 1M elements on the GPU
//   int blockSize = 256;
//   int numBlocks = (N + blockSize - 1) / blockSize;

//   add<<<numBlocks, blockSize>>>(N, x, y);

//   // Wait for GPU to finish before accessing on host
//   hipDeviceSynchronize();

//   // Check for errors (all values should be 3.0f)
//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = std::fmax(maxError, fabs(y[i]-3.0f));
//   cout << "Max error: " << maxError << std::endl;

//   // Free memory
//   hipFree(x);
//   hipFree(y);
}