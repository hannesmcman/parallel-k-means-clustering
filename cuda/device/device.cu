#include "hip/hip_runtime.h"
__device__
float euclidean_distance_array(const float * x,const float * y, int n) {
    float sum = 0;
    for (int i=0; i < n; i++) {
      sum += pow(x[i] - y[i], 2);
    }
    return sqrt(sum);
}

__device__
int find_nearest_center(int k, const float * features, int dimensions,float ** cluster){
    float minDist = FLT_MAX;
    int minIndex = 0;
    float dist = 0;
    for (int i=0; i<k; i++){
        dist = euclidean_distance_array(features, cluster[i], dimensions);
        if (dist < minDist) {
            minDist = dist;
            minIndex = i;
        }
    }
    return minIndex;
}

