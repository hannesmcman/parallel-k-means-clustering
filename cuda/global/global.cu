#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include "../device/device.cu"

// cluster assignment using randomization
__global__
void init_cluster_assignment(int k, int size, int * cluster_size, int * cluster_assignment){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    int group = 0;

    // srand(static_cast<unsigned int>(clock()));
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int random = index % k ;

    // int random = hiprandGenerate();
    // std::printf("block id : %d \t block dim: %d \t thread id : %d \t index : %d \t random : %d \n",blockIdx.x, blockDim.x, threadIdx.x, index, random);

    // cluster_assignment = new int[size];
    for (int i=index; i<size; i+=stride){
      group = (int) random;
    //   group = (int) (rand() % k);
      cluster_assignment[i] = group;
      cluster_size[group] += 1;
    }
  }

__global__
void update_clusters(int k, float ** cluster, const int * cluster_assignment, int data_size, \
                int dimensions, float  ** feature_vector,const int * cluster_size, int * response){
    response[0] = 0;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float ** temp;
    temp = new float* [k];
    for (int i=0; i<k; i++)
        temp[i] = new float[dimensions];
        for (int j=0; j<dimensions; j++){
            temp[i][j] = (float) 0;
        }
    }

    for (int i=index; i<data_size; i+=stride){
        for (int j=0; j<dimensions; j++){
        temp[cluster_assignment[i]][j] += feature_vector[i][j];   
        }
    }

    for (int i=0; i<k; i++){
        if (cluster_size[i] == 0){
//            std::printf("ZERO ::: %d \n", i);    
        continue;
        }
        for (int j=0; j<dimensions; j++){
        if (cluster[i][j] != temp[i][j]/cluster_size[i]){
            response[0] = 1;
        }
        cluster[i][j] = temp[i][j]/cluster_size[i];
        }
    }
}


__global__
void update_cluster_assignment(int k, int * cluster_assignment, int * cluster_size, float ** cluster, int size, int dimension, float ** features){
    for (int i=0; i<k; i++){
        cluster_size[i] = 0;
    }
        
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i<size; i+=stride){
        cluster_assignment[i] = find_nearest_center(k, features[i], dimension, cluster);
        cluster_size[cluster_assignment[i]]++;
    }
}