#include "hip/hip_runtime.h"
#include <float.h>
#include <cstdlib>
#include "../device/device.cu"

// __global__ void
// reduce0(float* g_idata,float* g_odata, unsigned int n) {
//     extern __shared__ float temp[];
//     int thid = threadIdx.x;
//     temp[thid] = g_idata[thid];
//     __syncthreads();
//     for(int offset = 1;offset < n; offset *= 2) {
//         if(thid >= offset)
//             temp[thid] += temp[thid - offset];
//         __syncthreads();
//     }
//     g_odata[thid] = temp[thid];
// }

// cluster assignment using randomization
__global__
void init_cluster_assignment(int k, int size, int * cluster_size, int * cluster_assignment){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    int group = 0;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int random = index % k ;

    for (int i=index; i<size; i+=stride){
      group = (int) random;
      cluster_assignment[i] = group;
      cluster_size[group] += 1;
    }
  }

__global__
void update_clusters(int k, float ** cluster, const int * cluster_assignment, int data_size, \
                int dimensions, float  ** feature_vector,const int * cluster_size, int * response){
    response[0] = 0;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    float ** temp;
    temp = new float* [k];
    for (int i=0; i<k; i++){
        temp[i] = new float[dimensions];
        for (int j=0; j<dimensions; j++){
            temp[i][j] = (float) 0;
        }
    }

    for (int i=0; i<data_size; i++){
        for (int j=0; j<dimensions; j++){
        temp[cluster_assignment[i]][j] += feature_vector[i][j];   
        }
    }

    for (int i=index; i<k; i+=stride){
        if (cluster_size[i] == 0){
            continue;
        }
        for (int j=0; j<dimensions; j++){
        if (cluster[i][j] != temp[i][j]/cluster_size[i]){
            response[0] = 1;
        }
        cluster[i][j] = temp[i][j]/cluster_size[i];
        }
    }
}


__global__
void update_cluster_assignment(int k, int * cluster_assignment, float ** cluster, int size, int dimension, float ** features){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i<size; i+=stride){
        cluster_assignment[i] = find_nearest_center(k, features[i], dimension, cluster);
    }
}