#include "hip/hip_runtime.h"
#include <float.h>
#include <string>
#include "../../lib/types.cpp"
#include "../cuda_lib/helpers.cu"
#include "../global/global.cu"

void calculate_cluster_size(int k, int *cluster_assignment,int n, int * cluster_size){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    for (int i=0; i<n; i++){
      cluster_size[cluster_assignment[i]]++;
    }
  }
  
void parse_data(const data_map &data, int &size, int &dimensions, string ** data_title, float *** data_features){
    vector<float> sample_map_data = data.begin()->second; 
    size = data.size();   
    dimensions = sample_map_data.size();

    hipMallocManaged(data_title, size*sizeof(string)); 
    hipMallocManaged(data_features, size*sizeof(float*));
    float ** data_v = *data_features;

    for (int i=0; i<size; i++)
        hipMallocManaged(&data_v[i], dimensions*sizeof(float)); 

    int index = 0;
    for (data_map::const_iterator it = data.begin(); it != data.end(); it++) {
        (*data_title)[index] = it->first;
        for (int j=0; j<dimensions; j++)
            data_v[index][j] = (it->second)[j];
        index++;
    } 
}

void init_clusters(int k, int size, int dimensions, float ** features, float ** cluster){
    srand(static_cast<unsigned int>(clock()));
    for (int i=0; i<k; i++){
        int index = (int) rand() % size;
        for (int j=0; j<dimensions; j++){
            cluster[i][j] = features[index][j];
        }
    }
}


int * find_clusters(int k, const data_map data, int max_iter) {
    // int iter = 0;
    int * cluster_size;
    int * cluster_assignment;

    // college data parsing
    int data_dimensions;
    int data_size;
    string * data_title;
    float ** data_features;
    parse_data(data, data_size, data_dimensions, &data_title, &data_features);

    hipMallocManaged(&cluster_size, k*sizeof(int));
    hipMallocManaged(&cluster_assignment, data_size*sizeof(int));

    int blockSize = 256;
    int numBlocks = (data_size + blockSize - 1) / blockSize;

    // init_cluster_assignment<<<numBlocks ,blockSize >>>(k, data_size, cluster_size, cluster_assignment);
    // Wait for GPU to finish before accessing on host
    // hipDeviceSynchronize();

    // calculate_cluster_size(k, cluster_assignment, data_size, cluster_size);

    float ** cluster;
    hipMallocManaged(&cluster, k*sizeof(float*));

    for (int i=0; i<k; i++)
        hipMallocManaged(&cluster[i], data_dimensions*sizeof(float));

    int * did_change;
    hipMallocManaged(&did_change, sizeof(int));

    //make clusters
    init_clusters(k, data_size, data_dimensions,data_features, cluster);
    update_cluster_assignment<<<numBlocks ,blockSize>>>(k, cluster_assignment, cluster_size, cluster, data_size, data_dimensions, data_features);
    hipDeviceSynchronize();    

    calculate_cluster_size(k, cluster_assignment, data_size, cluster_size); 

    for (int i=0; i < max_iter; i++) {
        update_clusters<<<numBlocks ,blockSize>>>(k, cluster, cluster_assignment, data_size, data_dimensions, data_features, cluster_size, did_change);
        hipDeviceSynchronize();

        if (did_change[0] == 1){
            // update_cluster_assignment(k, cluster_assignment, cluster_size, cluster, data);
            update_cluster_assignment<<<numBlocks ,blockSize>>>(k, cluster_assignment, cluster_size, cluster, data_size, data_dimensions, data_features);
            hipDeviceSynchronize();
                }
        else{    
            print_cluster_size(k, cluster_assignment,data_size);
            return cluster_assignment;
        }
            }

    print_cluster_size(k, cluster_assignment,data_size);
    return cluster_assignment;
}
