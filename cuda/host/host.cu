#include "hip/hip_runtime.h"
#include <float.h>
#include <iostream>
#include <string>
#include "../../lib/types.cpp"
#include "../cuda_lib/helpers.cu"
#include "../global/global.cu"
using namespace std;

void calculate_cluster_size(int k, int *cluster_assignment,int n, int * cluster_size){
    for (int i=0; i<k; i++)
      cluster_size[i] = 0;
    for (int i=0; i<n; i++){
      cluster_size[cluster_assignment[i]]++;
    }
  }
  
  void parse_data(const data_map &data, int &size, int &dimensions, string ** data_title, float *** data_features){
    vector<float> sample_map_data = data.begin()->second; 
    size = data.size();   
    dimensions = sample_map_data.size();

    hipMallocManaged(data_title, size*sizeof(string)); 
    hipMallocManaged(data_features, size*sizeof(float*));
    float ** data_v = *data_features;

    for (int i=0; i<size; i++)
        hipMallocManaged(&data_v[i], dimensions*sizeof(float)); 

    int index = 0;
    for (data_map::const_iterator it = data.begin(); it != data.end(); it++) {
        (*data_title)[index] = it->first;
        for (int j=0; j<dimensions; j++)
            data_v[index][j] = (it->second)[j];
        index++;
    } 
}

//initializing using Forgy
void init_clusters(int k, int size, int dimensions, float ** features, float ** cluster){
    srand(static_cast<unsigned int>(clock()));
    for (int i=0; i<k; i++){
        int index = (int) rand() % size;
        for (int j=0; j<dimensions; j++){
            cluster[i][j] = features[index][j];
        }
    }
}


int * find_clusters(int k, const data_map data, int max_iter) {
    // int iter = 0;
    int * cluster_size;
    int * cluster_assignment;

    // college data parsing
    int data_dimensions;
    int data_size;
    string * data_title;
    float ** data_features;
    parse_data(data, data_size, data_dimensions, &data_title, &data_features);
    cout << "parsing complete ..." << endl;

    hipMallocManaged(&cluster_size, k*sizeof(int));
    hipMallocManaged(&cluster_assignment, data_size*sizeof(int));

    int blockSize = 256;
    int numBlocks = (data_size + blockSize - 1) / blockSize;

    // init_cluster_assignment<<<numBlocks ,blockSize >>>(k, data_size, cluster_size, cluster_assignment);
    // Wait for GPU to finish before accessing on host
    // hipDeviceSynchronize();

    // calculate_cluster_size(k, cluster_assignment, data_size, cluster_size);

    float ** cluster;
    hipMallocManaged(&cluster, k*sizeof(float*));

    for (int i=0; i<k; i++)
        hipMallocManaged(&cluster[i], data_dimensions*sizeof(float));

    int * did_change;
    hipMallocManaged(&did_change, sizeof(int));

    //make clusters
    init_clusters(k, data_size, data_dimensions,data_features, cluster);
    cout << "Initialized cluster " << endl;

    update_cluster_assignment<<<numBlocks ,blockSize>>>(k, cluster_assignment, cluster, data_size, data_dimensions, data_features);
    hipDeviceSynchronize();    
    cout << "Updated assignment " << endl;
    calculate_cluster_size(k, cluster_assignment, data_size, cluster_size); 

    int iter =0;

    for (int i=0; i < max_iter; i++) {
        cout << "iteration :: " << iter++ << endl;

        update_clusters<<<numBlocks ,blockSize>>>(k, cluster, cluster_assignment, data_size, data_dimensions, data_features, cluster_size, did_change);
        hipDeviceSynchronize();

        cout << "iter: updated cluster " << endl;
        print_cluster_size(k, cluster_assignment,data_size);

        if (did_change[0] == 1){
            // update_cluster_assignment(k, cluster_assignment, cluster_size, cluster, data);
            update_cluster_assignment<<<numBlocks ,blockSize>>>(k, cluster_assignment, cluster, data_size, data_dimensions, data_features);
            hipDeviceSynchronize();

            calculate_cluster_size(k, cluster_assignment, data_size, cluster_size); 

            cout << "iter: updated assignment " << endl;

            }
        else{    
            print_cluster_size(k, cluster_assignment,data_size);
            return cluster_assignment;
        }
    }

    print_cluster_size(k, cluster_assignment,data_size);
    return cluster_assignment;
}
